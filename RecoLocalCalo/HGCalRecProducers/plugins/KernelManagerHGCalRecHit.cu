#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include "RecoLocalCalo/HGCalRecProducers/plugins/KernelManagerHGCalRecHit.h"
#include "HGCalRecHitKernelImpl.cuh"

KernelManagerHGCalRecHit::KernelManagerHGCalRecHit()
{
  ::nblocks_ = (data_->nhits_ + ::nthreads_.x - 1) / ::nthreads_.x;
}

KernelManagerHGCalRecHit::KernelManagerHGCalRecHit(KernelModifiableData<HGCUncalibratedRecHitSoA, HGCRecHitSoA> *data):
  data_(data)
{
  ::nblocks_ = (data_->nhits_ + ::nthreads_.x - 1) / ::nthreads_.x;
  nbytes_host_ = (data_->h_out_)->nbytes_ * data_->stride_;
  nbytes_device_ = (data_->d_1_)->nbytes_ * data_->stride_;
}

KernelManagerHGCalRecHit::~KernelManagerHGCalRecHit()
{
}

void KernelManagerHGCalRecHit::transfer_soas_to_device_()
{
  cudaCheck( hipMemcpyAsync((data_->d_1_)->amplitude_, (data_->h_in_)->amplitude_, nbytes_device_, hipMemcpyHostToDevice) );
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::transfer_soa_to_host_and_synchronize_()
{
  cudaCheck( hipMemcpyAsync((data_->h_out_)->energy_, (data_->d_out_)->energy_, nbytes_host_, hipMemcpyDeviceToHost) );
  cudaCheck( hipGetLastError() );
}

void KernelManagerHGCalRecHit::reuse_device_pointers_()
{
  std::swap(data_->d_1_, data_->d_2_); 
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGCeeUncalibratedRecHitConstantData> *kcdata)
{
  transfer_soas_to_device_();
  /*
  ee_step1<<<::nblocks_, ::nthreads_>>>( *(data_->d_2_), *(data_->d_1_), kcdata->data_, data_->nhits_ );
  after_();
  reuse_device_pointers_();
  */

  ee_to_rechit<<<::nblocks_, ::nthreads_>>>( *(data_->d_out_), *(data_->d_1_), kcdata->data_, data_->nhits_ );
  cudaCheck( hipGetLastError() );

  transfer_soa_to_host_and_synchronize_();
  cudaCheck( hipGetLastError() );
  cudaCheck( hipDeviceSynchronize() );
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGChefUncalibratedRecHitConstantData> *kcdata, const hgcal_conditions::HeterogeneousHEFConditionsESProduct* d_conds)
{
  transfer_soas_to_device_();
  cudaCheck( hipGetLastError() );

  /*
  hef_step1<<<::nblocks_,::nthreads_>>>( *(data_->d_2), *(data_->d_1_), d_kcdata->data, data_->nhits_);
  cudaCheck( hipGetLastError() );
  reuse_device_pointers_();
  */

  hef_to_rechit<<<::nblocks_,::nthreads_>>>( *(data_->d_out_), *(data_->d_1_), kcdata->data_, d_conds, data_->nhits_ );
  cudaCheck( hipGetLastError() );

  transfer_soa_to_host_and_synchronize_();
  cudaCheck( hipGetLastError() );
  cudaCheck( hipDeviceSynchronize() );
}

void KernelManagerHGCalRecHit::run_kernels(const KernelConstantData<HGChebUncalibratedRecHitConstantData> *kcdata)
{
  transfer_soas_to_device_();
  cudaCheck( hipGetLastError() );

  /*
  heb_step1<<<::nblocks_, ::nthreads_>>>( *(data_->d_2_), *(data_->d_1_), d_kcdata->data_, data_->nhits_);
  cudaCheck( hipGetLastError() );
  reuse_device_pointers_();
  */

  heb_to_rechit<<<::nblocks_, ::nthreads_>>>( *(data_->d_out_), *(data_->d_1_), kcdata->data_, data_->nhits_ );
  cudaCheck( hipGetLastError() );

  transfer_soa_to_host_and_synchronize_();
  cudaCheck( hipGetLastError() );
  cudaCheck( hipDeviceSynchronize() );
}

void KernelManagerHGCalRecHit::fill_positions(const hgcal_conditions::HeterogeneousHEFCellPositionsConditionsESProduct* d_conds)
{
  fill_positions_from_detids<<<::nblocks_,::nthreads_>>>(d_conds);
  cudaCheck( hipGetLastError() );
  
  //print_positions_from_detids<<<::nblocks_,::nthreads_>>>(d_conds);
}

HGCRecHitSoA* KernelManagerHGCalRecHit::get_output()
{
  return data_->h_out_;
}
