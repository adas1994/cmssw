#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include "DataFormats/ForwardDetId/interface/HGCalDetId.h"
#include "RecoLocalCalo/HGCalRecProducers/plugins/HGCalCellPositionsKernelImpl.cuh"

__global__ 
void fill_positions_from_detids(const hgcal_conditions::HeterogeneousHEFCellPositionsConditionsESProduct* conds)
{
  unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

  for (unsigned int i = tid; i < conds->nelems_posmap; i += blockDim.x * gridDim.x)
    {
      HeterogeneousHGCSiliconDetId did(conds->posmap.detid[i]);
      const float cU     = static_cast<float>( did.cellU()  );
      const float cV     = static_cast<float>( did.cellV()  );
      const float wU     = static_cast<float>( did.waferU() );
      const float wV     = static_cast<float>( did.waferV() );
      const float ncells = static_cast<float>( did.nCells() );
      const int32_t layer = did.layer();
      
      //based on `std::pair<float, float> HGCalDDDConstants::locateCell(const HGCSiliconDetId&, bool)
      const float r_x2 = conds->posmap.waferSize + conds->posmap.sensorSeparation;
      const float r = 0.5f * r_x2;
      const float sqrt3 = __fsqrt_rn(3.f);
      const float rsqrt3 = __frsqrt_rn(3.f); //rsqrt: 1 / sqrt
      const float R = r_x2 * rsqrt3;
      const float n2 = ncells / 2.f;
      const float yoff_abs = rsqrt3 * r_x2;
      const float yoff = (layer%2==1) ? yoff_abs : -1.f * yoff_abs; //CHANGE according to Sunanda's reply
      float xpos = (-2.f * wU + wV) * r;
      float ypos = yoff + (1.5f * wV * R);
      const float R1 = __fdividef( conds->posmap.waferSize, 3.f * ncells );
      const float r1_x2 = R1 * sqrt3;
      xpos += (1.5f * (cV - ncells) + 1.f) * R1;
      ypos += (cU - 0.5f * cV - n2) * r1_x2;

      conds->posmap.x[i] = xpos; //* side; multiply by -1 if one wants to obtain the position from the opposite endcap. CAREFUL WITH LATER DETECTOR ALIGNMENT!!!
      conds->posmap.y[i] = ypos;
    }
}

__global__
void print_positions_from_detids(const hgcal_conditions::HeterogeneousHEFCellPositionsConditionsESProduct* conds)
{
  unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

  for (unsigned int i = tid; i < conds->nelems_posmap; i += blockDim.x * gridDim.x)
    {
      HeterogeneousHGCSiliconDetId did(conds->posmap.detid[i]);
      const int32_t layer = did.layer();
      float posz = conds->posmap.zLayer[ layer-1 ];
      printf("PosX: %lf\t PosY: %lf\t Posz: %lf\n", conds->posmap.x[i], conds->posmap.y[i], posz);
    } 
}

//eventually this can also be written in parallel
__device__
unsigned map_cell_index(const float& cu, const float& cv, const unsigned& ncells)
{
  unsigned counter = 0;
  //left side of wafer
  for(int cellUmax=ncells, icellV=0; cellUmax<2*ncells && icellV<ncells; ++cellUmax, ++icellV)
    {
      for(int icellU=0; icellU<=cellUmax; ++icellU)
	{
	  if(cu == icellU and cv == icellV)
	    return counter;
	  else
	    counter += 1;
	}
    }
  //right side of wafer
  for(int cellUmin=1, icellV=ncells; cellUmin<=ncells && icellV<2*ncells; ++cellUmin, ++icellV)
    {
      for(int icellU=cellUmin; icellU<2*ncells; ++icellU)
	{
	  if(cu == icellU and cv == icellV)
	    return counter;
	  else
	    counter += 1;
	}
    }
  printf("ERROR: The cell was not found!");
  return 99;
}


//returns the index of the positions of a specific cell
__device__
unsigned hash_function(const int32_t& l, const float& wU, const float& wV, const float& cu, const float& cv, const hgcal_conditions::HeterogeneousHEFCellPositionsConditionsESProduct* conds)
{
  const unsigned thislayer = l - conds->posmap.firstLayer;
  const unsigned thisUwafer = wU - conds->posmap.waferMin;
  const unsigned thisVwafer = wV - conds->posmap.waferMin;
  const unsigned nwafers1D = conds->posmap.waferMax - conds->posmap.waferMin;

  //layer shift in terms of cell number
  unsigned ncells_up_to_thislayer = 0;
  for(unsigned q=0; q<thislayer; ++q)
    ncells_up_to_thislayer += conds->posmap.nCellsLayer[q];

  //waferU shift in terms of cell number
  unsigned ncells_up_to_thisUwafer = 0;
  unsigned waferU_shift = thislayer * nwafers1D;
  for(unsigned q=0; q<thisUwafer; ++q)
    ncells_up_to_thisUwafer += conds->posmap.nCellsWaferUChunk[waferU_shift + q];

  //waferV shift in terms of cell number
  unsigned ncells_up_to_thisVwafer = 0;
  const unsigned nwafers_up_to_thisLayer = thislayer * nwafers1D * nwafers1D;
  const unsigned nwafers_up_to_thisUwafer = thisUwafer * nwafers1D;
  for(unsigned q=0; q<thisVwafer; ++q)
      ncells_up_to_thisVwafer += conds->posmap.nCellsHexagon[nwafers_up_to_thisLayer + nwafers_up_to_thisUwafer + q];

  //cell shift in terms of cell number
  const unsigned ncells_thisVwafer = conds->posmap.nCellsHexagon[nwafers_up_to_thisLayer + nwafers_up_to_thisUwafer + thisVwafer];
  const unsigned cell_shift = map_cell_index(cu, cv, ncells_thisVwafer);

  const unsigned shift_total = ncells_up_to_thislayer + ncells_up_to_thisUwafer + ncells_up_to_thisVwafer + cell_shift;
  return shift_total;
}

__global__
void test(const unsigned& detid, const hgcal_conditions::HeterogeneousHEFCellPositionsConditionsESProduct* conds)
{
  unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

  if(tid == 0)
    {
      printf("Nelems: %u\n", static_cast<unsigned>(conds->nelems_posmap));
      for(unsigned i=0; i<1; ++i)
	{
	  /*
	  HeterogeneousHGCSiliconDetId did(detid);
	  const float cU     = static_cast<float>( did.cellU()  );
	  const float cV     = static_cast<float>( did.cellV()  );
	  const float wU     = static_cast<float>( did.waferU() );
	  const float wV     = static_cast<float>( did.waferV() );
	  const float ncells = static_cast<float>( did.nCells() );
	  const int32_t layer = did.layer();

	  const unsigned shift = hash_function(layer, wU, wV, cU, cV, conds);
	  printf("id: %d | x: %lf y: %lf\n", conds->posmap.detid[shift], conds->posmap.x[shift], conds->posmap.y[shift]);
	  */
	  printf("id: %d | x: %lf y: %lf\n", conds->posmap.detid[i], conds->posmap.x[i], conds->posmap.y[i]);
	}
    }
}
